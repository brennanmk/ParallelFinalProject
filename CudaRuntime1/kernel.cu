﻿
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>


__global__ void monteCarlo(long timeVal, int N, int a, int b, long double* answer)
{
	long double temp = 0;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	long double F; //variable for final integral
	int count = 0;



	hiprandState_t state;
	hiprand_init((timeVal * index), /* the seed controls the sequence of random values that are produced */
		0, /* the sequence number is only important with multiple cores */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&state);

	if (i < N) {
		int numGen = (hiprand(&state) % (b - a + 1)) + a;
		count++;
		double powerOf = (-1 * pow(numGen, 2) / 2);

		answer[i] = ((1 / sqrt(2 * 3.14)) * ((pow(2.718, powerOf))));

	}

}

int main() {
	int a = -5;
	int b = 5;
	int N = 100000000;
	long timeVal = time(NULL);
	double timeAvg;

	int size = N * sizeof(long double);

	long double* total = 0;
	long double* d_total;
	long double F = 0;
	long double temp = 0;


	clock_t startTime = clock(); //record start time (function found from https://en.cppreference.com/w/c/chrono/clock_t)

	total = (long double*)malloc(size);
	hipMalloc((void**)&d_total, size);

	int nblocks = (N + 511) / 512;


	hipMemcpy(d_total, total, size, hipMemcpyHostToDevice);

	monteCarlo <<<nblocks, 512 >>> (timeVal, N, a, b, d_total);

	hipMemcpy(total, d_total, size, hipMemcpyDeviceToHost);
	
	for (int i = 0; i < N; i++) {
		temp += total[i];

	}
	F = (((long double)b - (long double)a) / (long double)N) * temp;


	clock_t endTime = clock(); //end time (function found from https://en.cppreference.com/w/c/chrono/clock_t)
	timeAvg = ((double)(endTime - startTime)) / CLOCKS_PER_SEC; //add the elasped time to timeAvg


	printf("%lf\n", F);
	printf("Run Time = %fs\n", timeAvg); //Print average time

	hipFree(d_total);

	return 0;
}
