#include "hip/hip_runtime.h"
﻿
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

hipError_t addWithCuda(int N, int a, int b, long double *total);


__global__ void addKernel(int N, int a, int b, long double *total)
{
    long double temp = 0;
    int i = threadIdx.x;

    for (i = 0; i < N; i++) //perform provided calculations N times (add to total each time)
    {
        int numGen = (rand() % (b - a + 1)) + a;

        long double powerOf = (-1 * powl(numGen, 2) / 2);
        temp += ((1 / sqrtl(2 * 3.14)) * ((powl(2.718, powerOf))));
    }

    *total += temp;
}

int main(int argc, char* argv[])
{
    long double *total; //variable to store total
    int a = atoi(argv[1]);
    int b = atoi(argv[2]);
    int N = atoi(argv[3]);

    hipMallocManaged(&total, sizeof(long double));

    // Add vectors in parallel.
    addKernel << <1, 256 >> > (N, a, b, total);


    printf("%lf", total);

    return 0;
}
